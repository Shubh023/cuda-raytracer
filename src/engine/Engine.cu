#include "hip/hip_runtime.h"
#include <fstream>
#include <iostream>
#include <iomanip>
#include "hip/hip_runtime.h"

#include "Engine.h"
#include "object/Blob.h"
#include "image/Color.h"
#include "cmdline/MSG.h"

// Sample x & y positions for 4x AA
DEV static const float SAMPLE_4x[4 * 2] = {
    -1.0/4.0,  3.0/4.0,
     3.0/4.0,  1.0/3.0,
    -3.0/4.0, -1.0/4.0,
     1.0/4.0, -3.0/4.0,
};

// Sample x & y positions for 8x AA
DEV static const float SAMPLE_8x[8 * 2] = {
    -7.0f / 8.0f,  1.0f/ 8.0f,
    -5.0f / 8.0f, -5.0f / 8.0f,
    -1.0f / 8.0f, -3.0f / 8.0f,
     3.0f / 8.0f, -7.0f / 8.0f,
     5.0f / 8.0f, -1.0f / 8.0f,
     7.0f / 8.0f,  7.0f / 8.0f,
     1.0f / 8.0f,  3.0f / 8.0f,
    -3.0f / 8.0f,  5.0f / 8.0f
};

// Sample x & y positions for 16x AA
DEV static const float SAMPLE_16x[16 * 2] = {
    -8.0f / 8.0f,  0.0f / 8.0f,
    -6.0f / 8.0f, -4.0f / 8.0f,
    -3.0f / 8.0f, -2.0f / 8.0f,
    -2.0f / 8.0f, -6.0f / 8.0f,
     1.0f / 8.0f, -1.0f / 8.0f,
     2.0f / 8.0f, -5.0f / 8.0f,
     6.0f / 8.0f, -7.0f / 8.0f,
     5.0f / 8.0f, -3.0f / 8.0f,
     4.0f / 8.0f,  1.0f / 8.0f,
     7.0f / 8.0f,  4.0f / 8.0f,
     3.0f / 8.0f,  5.0f / 8.0f,
     0.0f / 8.0f,  7.0f / 8.0f,
    -1.0f / 8.0f,  3.0f / 8.0f,
    -4.0f / 8.0f,  6.0f / 8.0f,
    -7.0f / 8.0f,  8.0f / 8.0f,
    -5.0f / 8.0f,  2.0f / 8.0f
};

#define MAX_DEPTH 3

GBL void pixel_render(Color* colors, Scene* scene, int width, int height)
{
    int i = threadIdx.x + blockIdx.x * blockDim.x;
    int j = threadIdx.y + blockIdx.y * blockDim.y;
    if((i >= width) || (j >= height)) return;
    size_t pixel_index = j*width + i;
    auto cam = scene->get_camera();
    auto ray = cam->compute_ray(j,i);
    auto c = raytrace(*scene, ray, MAX_DEPTH);
    /*
    int samples = 16;
    float u = 0;
    float v = 0;
    for(int s=0; s < samples; s++){
        u = i + SAMPLE_16x[2 * s];
        v = j + SAMPLE_16x[2 * s + 1];
        ray = cam->compute_ray(v,u);
        c += raytrace(*scene, ray, MAX_DEPTH);
    }
    c /= float(samples + 1);
    */
    c.clamp();
    colors[pixel_index] = c;
}

Image& render(Scene* scene, Color* d_colors, int width, int height) {

    int nx = width;
    int ny = height;
    int tx = 8;
    int ty = 8;
    dim3 blocks(nx/tx+1,ny/ty+1);
    dim3 threads(tx,ty);
    pixel_render<<<blocks,threads>>>(d_colors, scene, nx,ny);
    checkCudaErrors(hipGetLastError());
    checkCudaErrors(hipDeviceSynchronize());

    Image* image = new Image(width, height);
    Color* colors = new Color[height * width];

    auto col1 = hipMemcpy(colors, d_colors, sizeof(Color) * width * height, hipMemcpyDeviceToHost);
    for (int j = height-1; j >= 0; j--) {
        for (int i = 0; i < width; i++) {
            size_t pixel_index = j*width + i;
            Color c = colors[pixel_index];
            // c.print();
            image->set_pixel(j,i,c);
        }
    }
    return *image;
}



DEV Color raytrace(Scene& scene, Ray &ray, int depth) {
    if (depth < 0)
        return scene.get_background(ray);

    nearest_object hit_pair = find_nearest_object(scene, ray);

    if (hit_pair.t == FLT_MAX or hit_pair.closest == nullptr)
        return scene.get_background(ray);

    Point3 hit_pos = ray.origin + ray.direction * hit_pair.t;
    Vector3 hit_normal = hit_pair.closest->normal(hit_pos);
    if (hit_normal % ray.direction > 0)
        hit_normal = -hit_normal;

    Vector3 reflected = ray.direction - (hit_normal * 2) * (ray.direction % hit_normal);
    MaterialParameter mp = hit_pair.closest->get_param_at(hit_pos);

    Color diffspec = get_color(scene, hit_pair.closest, hit_pos, hit_normal, reflected);
    Color reflection;

    if (mp.specularity) {
        auto mod_hit_pos = hit_pos + 0.001 * reflected; // Handle float approximation error
        Ray new_ray = Ray(mod_hit_pos, reflected);
        reflection += raytrace(scene, new_ray, depth - 1);
        reflection *= mp.specularity * mp.color;
    }
    Color refraction;
    if (mp.refractivity)
    {
        float etar = mp.refractivity;
        auto N = hit_normal.normalize();
        auto V = ray.direction.normalize();
        float first_comp = etar * (N.dot(V));
        float second_comp = std::sqrt(1 - std::pow(etar, 2) * (1 - std::pow(N.dot(V),2)));
        V3 refracted = (first_comp - second_comp) * N - etar * V;
        auto mod_hit_pos = hit_pos + 0.001 * refracted; // Handle float approximation error
        Ray T(mod_hit_pos, refracted);
        float kr = mp.transparency;
        refraction += kr * raytrace(scene,T, depth - 1);
    }
    return diffspec + reflection + refraction;
}


DEV nearest_object find_nearest_object(Scene& scene, Ray &ray) {
    float tmin = FLT_MAX;
    Object* closest = nullptr;

    for (int i = 0; i < scene.get_co(); i++) {
        Object* obj = scene.get_objects()[i];
        auto t = obj->intersect(ray);
        if (t > 0 && t < tmin) {
            tmin = t;
            closest = obj;
        }
    }
    nearest_object result = {};
    if (closest) {
        result.t = tmin;
        result.closest = closest;
        return result;
    }
    result.t = FLT_MAX;
    result.closest = nullptr;
    return result;
}

DEV Color get_color(Scene& scene, const Object* obj, Point3 pos, Vector3 normal, Vector3 reflected) {
    MaterialParameter mp = obj->get_param_at(pos);

    Color color;
    auto objects = scene.get_objects();
    for (int i = 0; i < scene.get_cl(); i++) {
        auto light = scene.get_lights()[i];

        bool hidden = false;
        for (auto j = 0; j < scene.get_co(); j++) {
            auto light_obj = objects[j];
            if (light_obj != obj) {
                auto light_direction = (light->get_origin() - pos);
                auto light_distance = light_direction.norm();
                Ray ray(pos, light_direction.normalize());
                auto t = light_obj->intersect(ray);
                if (t > 0 && t < light_distance) {
                    hidden = true;
                    break;
                }
            }
        }
        if (!hidden) {
            Vector3 to_light = light->get_direction(pos);
            float d_contribution = to_light % normal;
            float r_contribution = to_light % reflected;
            if (r_contribution > 0)
                r_contribution = std::pow(float(to_light % reflected), SPECULAR_NS);

            Color light_contrib = mp.color;

            if (d_contribution < 0)
                d_contribution = 0;
            light_contrib *= light->get_color(pos) * (d_contribution * mp.diffusion);

            if (r_contribution > 0)
                light_contrib += light->get_color(pos) * (r_contribution * mp.specularity);

            color += light_contrib;
        }
    }

    return color;
}
