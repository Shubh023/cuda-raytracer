#include "hip/hip_runtime.h"
//
// Created by shubh on 3/29/21.
//

#include "SpotLight.h"

DEV SpotLight::SpotLight(Point3 pos, Color c, double _intensity, const P3& _target, double innerA, double outerA)
        : Light(pos, c, _intensity), target(_target), innerAngle(innerA), outerAngle(outerA)
{}

DEV V3 SpotLight::get_direction(const P3 &p) const {
    return (get_origin() - p).normalize();
}


DEV  Color SpotLight::get_color(const P3 &p) const {
    V3 toSurface = (get_origin() - p).normalize();

    double angle = std::acos(toSurface.dot(get_direction(target)));
    if (angle > innerAngle + outerAngle) {
        return Color(0, 0, 0);
    }
    double diffuse = max(get_direction(target).dot(get_direction(p)), 0.1);
    diffuse *= (1.0 / toSurface.length());
    if (angle > innerAngle) {
        // double factor =  1 - ((angle - innerAngle) / outerAngle);
        double factor =  (angle - innerAngle) * (1 / (outerAngle - innerAngle));
        return ((Color(std::max(0.0, (color.r() / 255.0) * factor),
                     std::max(0.0, (color.g() / 255.0) * factor),
                     std::max(0.0, (color.b() / 255.0) * factor)) * intensity) * diffuse).clamp(0.0,1.0);
    }
    else
        return ((color * intensity) * diffuse).clamp(0.0,1.0);
}

DEV double SpotLight::get_intensity(const P3 &p) const {
    return intensity;
}



