#include "hip/hip_runtime.h"
//
// Created by shubh on 23/03/2021.
//

#include "Matrix33.h"
#include <err.h>
#include <stdio.h>

HD Vector3 M33::operator[](int i) const {
    switch (i) {
        case 0:
            return mat[0];
        case 1:
            return mat[1];
        case 2:
            return mat[2];
        default:
            printf("Matrix33 index out of bounds");
    }
    return mat[0];
}

HD Vector3 &Matrix33::operator[](int i) {
    switch (i) {
        case 0:
            return mat[0];
        case 1:
            return mat[1];
        case 2:
            return mat[2];
        default:
            printf("Matrix33 index out of bounds");
    };
    return mat[0];
}


HD Matrix33 Matrix33::T() {
    M33 id(1);
    for (int i = 0; i < 3; i++)
        for (int j = 0; j < 3; j++)
            id[i][j] = mat[j][i];
    return id;
}

HD V3 operator*(const M33& m, const V3& v)
{
    auto v0 = m[0];
    auto v1 = m[1];
    auto v2 = m[2];
    return Vector3(v.dot(v0),
                   v.dot(v1),
                   v.dot(v2));
}

HD V3 operator*(const V3& v, const M33& m)
{
    auto v0 = m[0];
    auto v1 = m[1];
    auto v2 = m[2];
    return Vector3(v.dot(v0),
                   v.dot(v1),
                   v.dot(v2));
}

HD M33 operator*(const M33& m1, const M33& m2)
{
    M33 mres(1);
    for (int i = 0; i < 3; i++) {
        for (int j = 0; j < 3; j++) {
            mres[i][j] = 0;
            for (int k = 0; k < 3; k++)
                mres[i][j] += m1[i][k] * m2[k][j];
        }
    }
    return mres;
    /*
    auto m2T = m2.T();
    V3 X = m1 * m2T[0];
    V3 Y = m1 * m2T[1];
    V3 Z = m1 * m2T[2];
    V3 W = m1 * m2T[3];
    return M33(X, Y, Z, W).T();
    */
}

HD bool operator==(const M33& m1, const M33& m2)
{
    return (m1[0] == m2[0]) && (m1[1] == m2[1]) && (m1[2] == m2[2]);
}

HD bool operator!=(const M33& m1, const M33& m2)
{
    return !(m1 == m2);
}

/**
 * Rotation Matrices
 */

HD M33 Matrix33::Rx(double theta)
{
    return M33(V3(1,0,0).normalized(),
               V3(0, cos(theta), -sin(theta)).normalized(),
               V3(0, sin(theta), cos(theta)).normalized());
}

HD M33 Matrix33::Ry(double theta)
{
    return M33(V3( cos(theta), 0, sin(theta)).normalized(),
               V3(0, 1,0).normalized(),
               V3(-sin(theta), 0, cos(theta)).normalized());
}

HD M33 Matrix33::Rz(double theta)
{
    return M33(V3( cos(theta), -sin(theta), 0).normalized(),
               V3(sin(theta), cos(theta), 0).normalized(),
               V3(0, 0,1).normalized());
}

HD M33 Matrix33::rotate(double alpha, double beta, double gamma)
{
    /**
     * alpha -> rotate around X axis
     * beta -> rotate around Y axis
     * gamma -> rotate around Z axis
     */
    return Rz(gamma) * Ry(beta) * Rx(alpha);
}

