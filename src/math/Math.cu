#include "hip/hip_runtime.h"
//
// Created by shubh on 23/03/2021.
//

#include "Math.h"

HD bool approximatelyeq(double x, double y)
{
    const double EPSILON = 1E-14;
    if (x == 0)
        return fabs(y) <= EPSILON;
    if (y == 0)
        return fabs(x) <= EPSILON;
    return fabs(x - y) / max(fabs(x), fabs(y)) <= EPSILON;
}