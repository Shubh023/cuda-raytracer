#include "hip/hip_runtime.h"
//
// Created by shubh on 23/03/2021.
//

#include "Vector3.h"
#include "../Macros.h"
#include <iostream>
#include <err.h>
#include <stdio.h>
/**
 * Basic Accessor & useful operators
 */
HD float& Vector3::operator[](int i)  {
    switch (i) {
        case 0:
            return _x;
        case 1:
            return _y;
        case 2:
            return _z;
        default:
            printf("Vector3 index out of bounds");
    }
    return _x;
}
/*
HD float &Vector3::operator[](int i) {
    switch (i) {
        case 0:
            return _x;
        case 1:
            return _y;
        case 2:
            return _z;
        default:
            warnx("Vector3 index out of bounds");
    }
    return 1;
}
*/


/**
  * Display Functions
  */

void Vector3::print() const {
    std::cout << "{ " << _x << ", " << _y << ", " << _z << " }" << std::endl;
}

std::ostream& operator<<(std::ostream &out, Vector3 &u)
{
    out << "{ " << u.x() << ", " << u.y() << ", " << u.z() << " }";
    return out;
}


/**
 * Enabling basic math operations between Vector3 and floats
 */


HD Vector3 operator*(const Vector3& a, const float d)
{
    return Vector3(a.x() * d, a.y() * d, a.z() * d);
}

HD Vector3 operator*(const float d, const Vector3& a)
{
    return Vector3(a.x() * d, a.y() * d, a.z() * d);
}

HD Vector3 operator/(const Vector3& a, const float d)
{
    return Vector3(a.x() / d, a.y() / d, a.z() / d);
}

HD Vector3 operator/(const float d, const Vector3& a)
{
    return Vector3(d / a.x(), d / a.y(), d / a.z());
}

HD Vector3 operator-(Vector3& a)
{
    return -1 * a;
}

/**
 * Enabling operations of type a [ + | - | * | / ] b
 */

HD Vector3 operator+(const Vector3& a, const Vector3& b)
{
    return Vector3(a.x() + b.x(), a.y() + b.y(), a.z() + b.z());
}

HD Vector3 operator-(const Vector3& a, const Vector3& b)
{
    return Vector3(a.x() - b.x(), a.y() - b.y(), a.z() - b.z());
}

HD Vector3 operator*(const Vector3& a, const Vector3& b)
{
    return Vector3(a.x() * b.x(), a.y() * b.y(), a.z() * b.z());
}

HD Vector3 operator/(const Vector3& a, const Vector3& b)
{
    return Vector3(a.x() / b.x(), a.y() / b.y(), a.z() / b.z());
}


/**
 * Enabling the assignation of values after operations of type a [ + | - | * | / ]= b
 */

HD Vector3& operator+=(Vector3& a, const Vector3& b)
{
    a.set(a.x() + b.x(), a.y() + b.y(), a.z() + b.z());
    return a;
}

HD Vector3& operator-=(Vector3& a, const Vector3& b)
{
    a.set(a.x() - b.x(), a.y() - b.y(), a.z() - b.z());
    return a;
}

HD Vector3& operator*=(Vector3& a, const Vector3& b)
{
    a.set(a.x() * b.x(), a.y() * b.y(), a.z() * b.z());
    return a;
}

HD Vector3& operator/=(Vector3& a, const Vector3& b)
{
    a.set(a.x() / b.x(), a.y() / b.y(), a.z() / b.z());
    return a;
}

HD Vector3& operator*=(Vector3& a, const float d)
{
    a.set(a.x() * d, a.y() * d, a.z() * d);
    return a;
}

HD Vector3& operator/=(Vector3& a, const float d)
{
    a.set(a.x() / d, a.y() / d, a.z() / d);
    return a;
}

/**
 * Classic Operations between Vector3 a and b
 */
HD float Vector3::dot(Vector3 u) const {
    return _x * u.x() + _y * u.y() + _z * u.z();
}

HD float Vector3::operator%(Vector3 u) const {
    return this->dot(u);
}

HD Vector3 Vector3::cross(Vector3 u) const {
    return Vector3(_y * u.z() - _z * u.y(),
                   _z * u.x() - _x * u.z(),
                   _x * u.y() - _y * u.x());
}

HD Vector3 Vector3::operator^(Vector3 u) const {
    return this->cross(u);
}

HD Vector3 Vector3::normalized() const {
    return *this / this->norm();
}

HD Vector3 Vector3::normalize() {
    this->set(*this / this->norm());
    return *this;
}

/*HD inline  Vector3 &Vector3::clamp(float min, float max) {
    auto x = std::max(std::min(float(_x), max), min);
    auto y = std::max(std::min(float(_y), max), min);
    auto z = std::max(std::min(float(_z), max), min);
    set(x,y,z);
    return *this;
}
 */


/**
 * Enabling Vector3 comparison and validation
 */

HD  bool operator==(const Vector3& a, const Vector3& b)
{
    return approximatelyeq(a._x, b._x) &&
           approximatelyeq(a._y, b._y) &&
           approximatelyeq(a._z, b._z);
}

HD  bool operator!=(const Vector3& a, const Vector3& b)
{
    return !(a == b);
}
