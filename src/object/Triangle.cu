#include "hip/hip_runtime.h"
#include "Triangle.h"

#define DOUBLE_EPSILON 1e-9
#define ABS_MIN 1e-9

// Handle floatequality comparison
DEV bool approximatelyEqual(float a, float b)
{
  if (a == b) return true;

  auto diff = std::abs(a-b);
  auto norm = std::min(std::abs(a + b), std::numeric_limits<float>::max());
  return diff < std::max(ABS_MIN, DOUBLE_EPSILON * norm);
}

DEV Triangle::Triangle(Point3 _a, Point3 _b, Point3 _c)
:a(_a), b(_b), c(_c) {
    AB = b - a;
    AC = c - a;
}

DEV float Triangle::intersect(const Ray& ray) {
    float det, inv_det;

    Vector3 p = ray.direction ^ AC;
    det = AB % p;

    if (approximatelyEqual(det, 0)) // Ray misses the plane
        return -1;

    inv_det = 1 / det;
    Vector3 OA = ray.origin - a;
    float u = (OA % p) * inv_det;
    if (u < 0.0 || u > 1.0)
        return -1;

    Vector3 q = OA ^ AB;
    float v = (ray.direction % q) * inv_det;
    if (v < 0.0 || u + v > 1.0)
        return -1;

    float t = (AC % q) * inv_det;
    return t;
}

DEV bool Triangle::bool_intersect(const Ray& ray) {
    float det, inv_det;

    Vector3 p = ray.direction ^ AC;
    det = AB % p;

    if (approximatelyEqual(det, 0))
        return false;
    if (det < 0) // Backface hit do not hide light
        return false;

    inv_det = 1 / det;
    Vector3 OA = ray.origin - a;
    float u = (OA % p) * inv_det;
    if (u < 0.0 || u > 1.0)
        return false;

    Vector3 q = OA ^ AB;
    float v = (ray.direction % q) * inv_det;
    if (v < 0.0 || u + v > 1.0)
        return false;

    return true;
}


DEV Vector3 Triangle::normal(const Point3& p) const {
    return (AB ^ AC).normalize();
}

DEV MaterialParameter Triangle::get_param_at(const P3 &point) const {
    return texmat->get_texmat(point);
}

DEV Color Triangle::get_color(const P3 &point) const {
    return texmat->get_color(point);
}
