#include "hip/hip_runtime.h"
//
// Created by shubh on 25/03/2021.
//

#include "Sphere.h"

DEV Sphere::Sphere(float x, float y, float z, float r) : Object(P3(x,y,z)), radius(r)
{}

DEV Sphere::Sphere(const P3 &o, float r) : Object(o), radius(r)
{}

DEV float Sphere::get_radius() const {
    return radius;
}

DEV void Sphere::set_radius(float r) {
    radius = r;
}

DEV float Sphere::intersect(const Ray &ray)
{
    V3 oc = ray.origin - origin;
    float b = 2 * oc.dot(ray.direction);
    float a = ray.direction.length();
    float c = oc.length() - radius*radius;
    float discriminant = b * b - 4 * a * c;
    if(discriminant < 0)
        return -1;
    float t = (-b - std::sqrt(discriminant)) / 2;
    if (t > 0)
        return t;
    return -1;
}

DEV bool Sphere::bool_intersect(const Ray &ray)
{
    return (intersect(ray) > 0);
}

DEV V3 Sphere::normal(const P3 &point) const {
    return (point - origin).normalized();
}

DEV MaterialParameter Sphere::get_param_at(const P3 &point) const {
    return texmat->get_texmat(point);
}

DEV Color Sphere::get_color(const P3 &point) const {
    return texmat->get_color(point);
}
