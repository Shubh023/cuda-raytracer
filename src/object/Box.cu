#include "hip/hip_runtime.h"
#include "Box.h"

DEV float Box::intersect(const Ray &ray) {
    float tmin, tmax, tymin, tymax, tzmin, tzmax;
    if (ray.direction.x() >= 0) {
        tmin = (min.x - ray.origin.x) / ray.direction.x();
        tmax = (max.x - ray.origin.x) / ray.direction.x();
    }
    else {
        tmin = (max.x - ray.origin.x) / ray.direction.x();
        tmax = (min.x - ray.origin.x) / ray.direction.x();
    }
    if (ray.direction.y() >= 0) {
        tymin = (min.y - ray.origin.y) / ray.direction.y();
        tymax = (max.y - ray.origin.y) / ray.direction.y();
    } else {
        tymin = (max.y - ray.origin.y) / ray.direction.y();
        tymax = (min.y - ray.origin.y) / ray.direction.y();
    }
    if ((tmin > tymax) || (tymin > tmax)) {
        return -1;
    }
    if (tymin > tmin) {
        tmin = tymin;
    }
    if (tymax < tmax) {
        tmax = tymax;
    }
    if (ray.direction.z() >= 0) {
        tzmin = (min.z - ray.origin.z) / ray.direction.z();
        tzmax = (max.z - ray.origin.z) / ray.direction.z();
    } else {
        tzmin = (max.z - ray.origin.z) / ray.direction.z();
        tzmax = (min.z - ray.origin.z) / ray.direction.z();
    }
    if ((tmin > tzmax) || (tzmin > tmax)) {
        return -1;
    }
    if (tzmin > tmin) {
        tmin = tzmin;
    }
    if (tzmax < tmax) {
        tmax = tzmax;
    }
    return tmin;
}


DEV bool Box::bool_intersect(const Ray &ray) {

    return (intersect(ray) > 0);
}

DEV V3 Box::normal(const P3 &point) const {
    V3 normal;
    P3 size(abs(min.x - max.x),
            abs(min.y - max.y),
            abs(min.z - max.z));
    V3 localpoint = (point - P3(0,0,0)) - (max - size * 0.5);

    float mini = std::numeric_limits<float>::max();
    float distance = std::abs(size.x - abs(localpoint.x()));
    if (distance < mini)
    {
        mini = distance;
        normal.set(1,0,0);
        int sign = (localpoint.x() < 0) ? -1 : 1;
        normal *= sign;
    }
    distance = std::abs(size.y - abs(localpoint.y()));
    if (distance < mini)
    {
        mini = distance;
        normal.set(0,1,0);
        int sign = (localpoint.y() < 0) ? -1 : 1;
        normal *= sign;
    }
    distance = std::abs(size.z - abs(localpoint.z()));
    if (distance < mini)
    {
        mini = distance;
        normal.set(0,0,1);
        int sign = (localpoint.z() < 0) ? -1 : 1;
        normal *= sign;
    }
    return normal.normalized();
}

DEV MaterialParameter Box::get_param_at(const P3 &point) const {
    return texmat->get_texmat(point);
}

DEV Color Box::get_color(const P3 &point) const {
    return texmat->get_color(point);
}


