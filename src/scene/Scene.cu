#include "Scene.h"
#include "hip/hip_runtime.h"

DEV Camera* Scene::get_camera() {
    return camera;
}

DEV void Scene::set_camera(int width, int height) {
    camera->set_resolution(width, height);
}

DEV Color Scene::get_background(Ray&) {
    return Color(0.0, 0.0, 0.0);
}

DEV void Scene::add_object(Object* object) {
    if (maxo > co)
        objects[co++] = object;
    /*
    else
        std::cerr << "Max Objects amount reached" << std::endl;
    */
}

DEV void Scene::add_light(Light* light) {
    if (maxl > cl)
        lights[cl++] = light;
    /*
    else
        std::cerr << "Max Lights amount reached" << std::endl;
        */
}

DEV Object** Scene::get_objects() {
    return objects;
}

DEV Light** Scene::get_lights() {
    return lights;
}
