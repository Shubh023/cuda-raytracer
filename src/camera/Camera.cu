#include "hip/hip_runtime.h"
//
// Created by shubh on 23/03/2021.
//

#include "Camera.h"

DEV Camera::Camera(P3 &_origin, P3 &_target, V3 &_up, float _alpha, float _beta, float _zmin) {

    /** Setting up Camera related Vectors **/
    origin = _origin;
    target = _target;
    up = _up;
    V3 front = (target - origin).normalized();
    V3 other = up.cross(front).normalized();

    /** Setting up Camera Coordinate System using Camera related Vectors **/
    // camera_matrix[0] = front.normalized();
    // camera_matrix[1] = up.normalized();
    // camera_matrix[2] = other.normalized();
    camera_matrix = M33(up,up,up);
    camera_matrix.set(front.normalized(), up.normalized(), other.normalized());
    zmin = _zmin;
    alpha = _alpha;
    beta = _beta;
    set_resolution(1920, 1080);
}

DEV void Camera::set_zmin(float _zmin) {
    zmin = _zmin;
}

DEV void Camera::set_resolution(int _width, int _height) {
    width = _width;
    height = _height;
    float real_width = std::tan(alpha / 2) * zmin;
    float real_height = std::tan(beta / 2) * zmin;

    plane_up = camera_matrix[1] * (2 * real_height / (_height - 1));
    plane_right = camera_matrix[2] * (2 * real_width / (_width - 1));
    plane_bottom_left = origin + (camera_matrix[0] * zmin) // Go to image plane
                        - (camera_matrix[1] * real_height) // Go to bottom border
                        - (camera_matrix[2] * real_width); // Go to left border
}

DEV Ray Camera::compute_ray(int x, int y) {
    Point3 pixel = plane_bottom_left + (x * plane_up) + (y * plane_right);
    Ray ray(origin, (pixel - origin).normalized());
    return ray;
}

DEV void Camera::rotate(float alpha, float beta, float gamma) {
    M33 rotation_matrix = camera_matrix.rotate(gamma, beta, alpha);
    camera_matrix = rotation_matrix * camera_matrix;
}
