#include "hip/hip_runtime.h"
//
// Created by shubh on 23/03/2021.
//

#include "Color.h"
#include <err.h>
#include <stdio.h>
/** METHODS for Color class **/

HD Color& Color::clamp(float min, float max) {
    auto r = std::max(std::min(float(_r), max), min);
    auto g = std::max(std::min(float(_g), max), min);
    auto b = std::max(std::min(float(_b), max), min);
    set(r,g,b);
    return *this;
}

HD float Color::operator[](int i) const {
    switch (i) {
        case 0:
            return _r;
        case 1:
            return _g;
        case 2:
            return _b;
        default:
            printf("Color index out of bounds");
            return {};
    }
    return {};
}

/*
HD float &Color::operator[](int i) {
    switch (i) {
        case 0:
            return _r;
        case 1:
            return _g;
        case 2:
            return _b;
        default:
            throw std::out_of_range ("Color index out of bounds");
    }
}
*/
void Color::print() const {
    std::cout << "{ " << _r << ", " << _g << ", " << _b << " }" << std::endl;
}


std::ostream& operator<<(std::ostream &out, Color &c)
{
    out << "{ " << c.r() << ", " << c.g() << ", " << c.b() << " }";
    return out;
}


/** METHODS for RGB class **/
uint8_t RGB::operator[](int i) const {
    switch (i) {
        case 0:
            return _r;
        case 1:
            return _g;
        case 2:
            return _b;
        default:
            throw std::out_of_range ("Color index out of bounds");
    }
}

uint8_t &RGB::operator[](int i) {
    switch (i) {
        case 0:
            return _r;
        case 1:
            return _g;
        case 2:
            return _b;
        default:
            throw std::out_of_range ("Color index out of bounds");
    }
}

void RGB::print() const {
    std::cout << "{ " << _r << ", " << _g << ", " << _b << " }" << std::endl;
}


std::ostream& operator<<(std::ostream &out, RGB &c)
{
    out << "{ " << c.r() << ", " << c.g() << ", " << c.b() << " }";
    return out;
}

HD Color operator+(const Color& c1, const Color& c2) {
    return Color(c1.r() + c2.r(), c1.g() + c2.g(), c1.b() + c2.b());
}

HD Color operator*(const Color& c1, const Color& c2) {
    return Color(c1.r() * c2.r(), c1.g() * c2.g(), c1.b() * c2.b());
}

HD Color operator-(const Color& c1, const Color& c2) {
    return Color(c1.r() - c2.r(), c1.g() - c2.g(), c1.b() - c2.b());
}

HD Color operator*(const Color& c, float f) {
    return Color(c.r() * f, c.g() * f, c.b() * f);
}

HD Color operator*(float f, const Color& c) {
    return Color(c.r() * f, c.g() * f, c.b() * f);
}

HD Color operator/(const Color& c, float f) {
    return Color(c.r() / f, c.g() / f, c.b() / f);
}

HD Color operator/(float f, const Color& c) {
    return Color(c.r() / f, c.g() / f, c.b() / f);
}

HD Color& operator+=(Color& c1, const Color& c2) {
    c1.set(c1.r() + c2.r(), c1.g() + c2.g(), c1.b() + c2.b());
    return c1;
}

HD Color& operator-=(Color& c1, const Color& c2) {
    c1.set(c1.r() - c2.r(), c1.g() - c2.g(), c1.b() - c2.b());
    return c1;
}

HD Color& operator*=(Color& c, float f) {
    c.set(c.r() * f, c.g() * f, c.b() * f);
    return c;
}

HD Color& operator/=(Color& c, float f) {
    c.set(c.r() / f, c.g() / f, c.b() / f);
    return c;
}

HD Color& operator*=(Color& c1, const Color& c2) {
    c1.set(c1.r() * c2.r(), c1.g() * c2.g(), c1.b() * c2.b());
    return c1;
}
