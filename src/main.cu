#include "hip/hip_runtime.h"
#include <iostream>
#include "hip/hip_runtime.h"

#include "Macros.h"
#include "math/utils.h"
#include "image/Image.h"
#include "engine/Engine.h"
#include "engine/Engine.h"
#include "light/PointLight.h"
#include "light/DirectionalLight.h"
#include "light/SpotLight.h"
#include "object/Sphere.h"
#include "object/Plane.h"
#include "object/Triangle.h"
#include "object/Box.h"
#include "object/Blob.h"
#include "object/SmoothTriangle.h"
#include "texture_material/ChequeredTexture.h"
#include "texture_material/ParametricTexture.h"

#define IMAGE_WIDTH 2200
#define IMAGE_HEIGHT 2000

void check_cud(hipError_t result, char const *const func, const char *const file, int const line) {
    if (result) {
        std::cerr << "CUDA error = " << static_cast<unsigned int>(result) << " at " <<
                  file << ":" << line << " '" << func << "' \n";
        // Make sure we call CUDA Device Reset before exiting
        hipDeviceReset();
        exit(99);
    }
}

DEV void cursedSpheres(Scene* scene, P3 origin, float rayon, int i)
{
    if (!i)
        return;
    auto sphere = new Sphere(origin, rayon);
    Color col(i * 0.25,i * 0.5,i * 0.75);
    auto texmat = new UniformTexture(col.clamp(0.f,1.f));
    texmat->set_diffusion(1);
    texmat->set_specularity(1);
    sphere->set_texmat(texmat);
    scene->add_object(sphere);
    int r = rayon * 1.25;
    cursedSpheres(scene, origin + P3(r,0,0), rayon * 1/2, i - 1);
    cursedSpheres(scene, origin + P3(-r,0,0), rayon * 1/2, i - 1);
    cursedSpheres(scene, origin + P3(0, r,0), rayon * 1/2, i - 1);
    cursedSpheres(scene, origin + P3(0, -r,0), rayon * 1/2, i - 1);
    cursedSpheres(scene, origin + P3(0,0,-r), rayon * 1/2, i - 1);
    cursedSpheres(scene, origin + P3(0,0,r), rayon * 1/2, i - 1);
}

GBL void spheresfractal(Scene* d_scene)
{

    uint width = IMAGE_WIDTH;
    uint height = IMAGE_HEIGHT;
    double alpha = 45 * M_PI / 180;
    double beta = std::atan(double(height * std::tan(alpha)) / width);

    // Camera
    Point3 cam_pos(0, 0, -90);
    Point3 cam_target(0, 0, 0);
    V3 up(0, 1, 0);

    Camera* cam = new Camera(cam_pos, cam_target, up, alpha, beta, 2);
    cam->rotate(0 * M_PI / 180,  0 * M_PI / 180, 0 * M_PI / 180);

    Scene* scene = new Scene(cam, 800, 10);
    // Colors
    Color red(1, 0, 0);
    Color green(0, 1, 0);
    Color blue(0, 0, 1);
    Color orange(1, 0.5, 0);
    Color white(1, 1, 1);
    Color black(0, 0, 0);

    // Textures
    // Texture Red
    auto TexRed = new UniformTexture(red);
    TexRed->set_diffusion(0.8);
    TexRed->set_specularity(0.2);
    TexRed->set_refractivity(1.0);
    // Texture Green
    auto TexGreen =new UniformTexture(green);
    TexGreen->set_diffusion(0.8);
    TexGreen->set_specularity(0.2);
    //TexGreen->set_refractivity(1.15);

    // Texture Blue
    auto TexBlue = new UniformTexture(blue);
    TexBlue->set_diffusion(0.8);
    TexBlue->set_specularity(0.2);
    //TexBlue->set_refractivity(1.15);
    // Texture Orange
    auto TexOrange = new UniformTexture(white);
    TexOrange->set_diffusion(1);
    TexOrange->set_specularity(0.1);
    // Texture White
    auto TexWhite = new UniformTexture(white);
    TexWhite->set_diffusion(0.1);
    TexWhite->set_specularity(0.0);
    TexWhite->set_refractivity(0.05);
    TexWhite->set_transparency(1.0);

    // Texture 2
    auto Cheq = new ChequeredBoardMaterial(white, black);
    Cheq->set_diffusion(0,1);
    Cheq->set_diffusion(1,0);
    Cheq->set_specularity(0,0.1);
    Cheq->set_specularity(1,0.0);

    // Lights
    // Lights
    Point3 l1(1, 1, -1);
    Vector3 direction = l1 - P3(0,0,0);
    auto light1 = new DirectionalLight(l1, white, 1, direction);
    scene->add_light(light1);

    Point3 l2(-1, 1, -1);
    Vector3 direction2 = l2 - P3(0,0,0);
    auto light2 = new DirectionalLight(l2, white, 1, direction2);
    scene->add_light(light2);

    Point3 l3(0, 0, -1);
    Vector3 direction3 = l2 - P3(0,0,0);
    auto light3 = new DirectionalLight(l3, white, 1, direction3);
    scene->add_light(light3);

    Point3 l4(0, 0, -5);
    auto light4 = new PointLight(l4, white, 20);
    // scene->add_light(light4);

    auto camlight = new SpotLight(cam->get_origin(), white, 0.5, cam->get_target(),
                                  60 * (M_PI/180),
                                  80 * (M_PI/180));
    scene->add_light(camlight);

    // Plane
    auto plane = new Plane(P3(0,0,10), Vector3(0,0,1));
    plane->set_texmat(TexRed);
    scene->add_object(plane);


    int size = 10;
    int rayon = 10;
    P3 origin(0, 0, 0);

    cursedSpheres(scene,origin,rayon, 4);

    scene->set_camera(width, height);
    *d_scene = *scene;
}


int main(int argc, char* argv[])
{

    int devNum = 0;
    checkCudaErrors(hipGetDevice(&devNum));
    checkCudaErrors(hipSetDevice(devNum));

    std::cout << "argc : " << argc << std::endl;
    uint width = IMAGE_WIDTH;
    uint height = IMAGE_HEIGHT;

    size_t size;
    hipDeviceGetLimit(&size, hipLimitStackSize);
    std::cout << "Stack size limit: " << size << "\n";
    checkCudaErrors(hipDeviceSetLimit(hipLimitStackSize, 102400));
    hipDeviceGetLimit(&size, hipLimitStackSize);
    std::cout << "New stack size limit: " << size << "\n";

    Color *d_colors;
    int col_size = sizeof(Color) * width * height;
    checkCudaErrors(hipMalloc((void**)&d_colors, col_size));
    Scene* scene;
    checkCudaErrors(hipMalloc((void**)&scene, sizeof(Scene)));

    spheresfractal<<<1,1>>>(scene);
    checkCudaErrors(hipGetLastError());
    checkCudaErrors(hipDeviceSynchronize());

    auto image = render(scene, d_colors, width, height);
    image.save("img.ppm");
    hipFree(d_colors);
    hipFree(scene);
    hipDeviceReset();

    return 0;
}
